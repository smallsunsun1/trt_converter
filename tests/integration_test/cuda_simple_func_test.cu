
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

int main(int argc, char* argv[]) {
    hipSetDevice(0);
    hipEvent_t start;
    hipEvent_t end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, end);
    std::cout << "elapsed_time: " << elapsed_time << std::endl;
}